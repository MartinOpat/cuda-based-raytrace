#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <cmath>

#include "linalg/linalg.h"
#include "objs/sphere.h"
#include "img/handler.h"

#define WIDTH 3840
#define HEIGHT 2160
#define SAMPLES_PER_PIXEL 8


__device__ Vec3 phongShading(const Vec3& point, const Vec3& normal, const Vec3& lightDir, const Vec3& viewDir, const Vec3& color) {
    double ambientStrength = 0.1;
    double diffuseStrength = 0.8;
    double specularStrength = 0.5;
    int shininess = 64;

    Vec3 ambient = color * ambientStrength;
    double diff = max(normal.dot(lightDir), 0.0);
    Vec3 diffuse = color * (diffuseStrength * diff);

    Vec3 reflectDir = (normal * (2.0 * normal.dot(lightDir)) - lightDir).normalize();
    double spec = pow(max(viewDir.dot(reflectDir), 0.0), shininess);
    Vec3 specular = Vec3(1.0, 1.0, 1.0) * (specularStrength * spec);

    return ambient + diffuse + specular;
}

__global__ void renderKernel(unsigned char* framebuffer, Sphere* spheres, int numSpheres, Vec3 lightPos) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    int pixelIndex = (y * WIDTH + x) * 3;
    Vec3 rayOrigin(0, 0, 0);
    Vec3 colCum(0, 0, 0);

    double spp = static_cast<double>(SAMPLES_PER_PIXEL);
    for (int sample = 0; sample < SAMPLES_PER_PIXEL; sample++) {
        double u = (x + (sample / spp) - WIDTH / 2.0) / WIDTH;
        double v = (y + (sample / spp) - HEIGHT / 2.0) / HEIGHT;
        Vec3 rayDir(u, v, 1.0);
        rayDir = rayDir.normalize();

        for (int i = 0; i < numSpheres; ++i) {
            double t;
            if (spheres[i].intersect(rayOrigin, rayDir, t)) {
                Vec3 hitPoint = rayOrigin + rayDir * t;
                Vec3 normal = (hitPoint - spheres[i].center).normalize();
                Vec3 lightDir = (lightPos - hitPoint).normalize();
                Vec3 viewDir = -rayDir;

                colCum = colCum + phongShading(hitPoint, normal, lightDir, viewDir, spheres[i].color);
            }
        }
    }

    // Average color across all samples
    Vec3 color = colCum * (1.0 / SAMPLES_PER_PIXEL);

    framebuffer[pixelIndex] = static_cast<unsigned char>(fmin(color.x, 1.0) * 255);
    framebuffer[pixelIndex + 1] = static_cast<unsigned char>(fmin(color.y, 1.0) * 255);
    framebuffer[pixelIndex + 2] = static_cast<unsigned char>(fmin(color.z, 1.0) * 255);
}



int main() {
    Sphere spheres[] = {
        { Vec3(0, 0, 5), 1.0, Vec3(1.0, 0.0, 0.0) },  // Red sphere
        { Vec3(-2, 1, 7), 1.0, Vec3(0.0, 1.0, 0.0) }, // Green sphere
        { Vec3(2, -1, 6), 1.0, Vec3(0.0, 0.0, 1.0) }  // Blue sphere
    };
    int numSpheres = sizeof(spheres) / sizeof(Sphere);
    Vec3 lightPos(5, 5, 0);

    unsigned char* d_framebuffer;
    unsigned char* h_framebuffer = new unsigned char[WIDTH * HEIGHT * 3];
    Sphere* d_spheres;
    hipMalloc(&d_framebuffer, WIDTH * HEIGHT * 3);
    hipMalloc(&d_spheres, numSpheres * sizeof(Sphere));
    hipMemcpy(d_spheres, spheres, numSpheres * sizeof(Sphere), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);
    renderKernel<<<numBlocks, threadsPerBlock>>>(d_framebuffer, d_spheres, numSpheres, lightPos);
    hipDeviceSynchronize();

    hipMemcpy(h_framebuffer, d_framebuffer, WIDTH * HEIGHT * 3, hipMemcpyDeviceToHost);
    saveImage("output.ppm", h_framebuffer, WIDTH, HEIGHT);

    hipFree(d_framebuffer);
    hipFree(d_spheres);
    delete[] h_framebuffer;

    std::cout << "High-resolution image saved as output.ppm" << std::endl;
    return 0;
}
