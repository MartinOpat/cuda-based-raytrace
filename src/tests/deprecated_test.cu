#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>

#define WIDTH 800
#define HEIGHT 600

struct Vec3 {
    double x, y, z;

    __host__ __device__ Vec3() : x(0), y(0), z(0) {}
    __host__ __device__ Vec3(double x, double y, double z) : x(x), y(y), z(z) {}

    __host__ __device__ Vec3 operator+(const Vec3& b) const { return Vec3(x + b.x, y + b.y, z + b.z); }
    __host__ __device__ Vec3 operator-(const Vec3& b) const { return Vec3(x - b.x, y - b.y, z - b.z); }
    __host__ __device__ Vec3 operator*(double b) const { return Vec3(x * b, y * b, z * b); }
    __host__ __device__ Vec3 operator-() const { return Vec3(-x, -y, -z); }
    __host__ __device__ double dot(const Vec3& b) const { return x * b.x + y * b.y + z * b.z; }
    __host__ __device__ Vec3 normalize() const { double len = sqrt(x * x + y * y + z * z); return Vec3(x / len, y / len, z / len); }
};

// Simple Phong lighting components
struct Sphere {
    Vec3 center;
    double radius;
    Vec3 color;

    __device__ bool intersect(const Vec3& rayOrigin, const Vec3& rayDir, double& t) const {
        Vec3 oc = rayOrigin - center;
        double b = oc.dot(rayDir);
        double c = oc.dot(oc) - radius * radius;
        double h = b * b - c;
        if (h < 0.0) return false;
        h = sqrt(h);
        t = -b - h;
        return true;
    }
};

__device__ Vec3 phongShading(const Vec3& point, const Vec3& normal, const Vec3& lightDir, const Vec3& viewDir, const Vec3& color) {
    double ambientStrength = 0.1;
    double diffuseStrength = 0.8;
    double specularStrength = 0.5;
    int shininess = 32;

    // Ambient
    Vec3 ambient = color * ambientStrength;

    // Diffuse
    double diff = max(normal.dot(lightDir), 0.0);
    Vec3 diffuse = color * (diffuseStrength * diff);

    // Specular
    Vec3 reflectDir = (normal * (2.0 * normal.dot(lightDir)) - lightDir).normalize();
    double spec = pow(max(viewDir.dot(reflectDir), 0.0), shininess);
    Vec3 specular = Vec3(1.0, 1.0, 1.0) * (specularStrength * spec);

    return ambient + diffuse + specular;
}

__global__ void renderKernel(unsigned char* framebuffer, Sphere sphere, Vec3 lightPos) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    int pixelIndex = (y * WIDTH + x) * 3;

    Vec3 rayOrigin(0, 0, 0);
    Vec3 rayDir((x - WIDTH / 2.0) / WIDTH, (y - HEIGHT / 2.0) / HEIGHT, 1.0);
    rayDir = rayDir.normalize();

    double t;
    if (sphere.intersect(rayOrigin, rayDir, t)) {
        Vec3 hitPoint = rayOrigin + rayDir * t;
        Vec3 normal = (hitPoint - sphere.center).normalize();
        Vec3 lightDir = (lightPos - hitPoint).normalize();
        Vec3 viewDir = -rayDir;

        Vec3 color = phongShading(hitPoint, normal, lightDir, viewDir, sphere.color);

        framebuffer[pixelIndex] = static_cast<unsigned char>(fmin(color.x, 1.0) * 255);
        framebuffer[pixelIndex + 1] = static_cast<unsigned char>(fmin(color.y, 1.0) * 255);
        framebuffer[pixelIndex + 2] = static_cast<unsigned char>(fmin(color.z, 1.0) * 255);
    } else {
        framebuffer[pixelIndex] = 0;
        framebuffer[pixelIndex + 1] = 0;
        framebuffer[pixelIndex + 2] = 0;
    }
}

void saveImage(const char* filename, unsigned char* framebuffer) {
    std::ofstream imageFile(filename, std::ios::out | std::ios::binary);
    imageFile << "P6\n" << WIDTH << " " << HEIGHT << "\n255\n";
    for (int i = 0; i < WIDTH * HEIGHT * 3; i++) {
        imageFile << framebuffer[i];
    }
    imageFile.close();
}

int main() {
    // Initialize sphere and light source
    Sphere sphere = { Vec3(0, 0, 5), 1.0, Vec3(1.0, 0.0, 0.0) };  // Red sphere
    Vec3 lightPos(5, 5, 0);

    // Allocate framebuffer on device and host
    unsigned char* d_framebuffer;
    unsigned char* h_framebuffer = new unsigned char[WIDTH * HEIGHT * 3];
    hipMalloc(&d_framebuffer, WIDTH * HEIGHT * 3);

    // Launch
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);
    renderKernel<<<numBlocks, threadsPerBlock>>>(d_framebuffer, sphere, lightPos);
    hipDeviceSynchronize();

    // Copy result back to host and save
    hipMemcpy(h_framebuffer, d_framebuffer, WIDTH * HEIGHT * 3, hipMemcpyDeviceToHost);
    saveImage("output.ppm", h_framebuffer);

    // Clean up
    hipFree(d_framebuffer);
    delete[] h_framebuffer;

    std::cout << "Image saved as output.ppm" << std::endl;
    return 0;
}
