#include "FrameBuffer.h"
#include "linalg/linalg.h"


__host__ FrameBuffer::FrameBuffer(unsigned int w, unsigned int h) : w(w), h(h) {
  this->buffer_size = w*h*sizeof(unsigned int);
  hipMalloc((void**)&this->buffer, this->buffer_size);
  hipMemset(this->buffer, 0, this->buffer_size);
}

__device__ void FrameBuffer::writePixel(int x, int y, float r, float g, float b) {
  int i = y * this->w + x;

  // the opengl buffer uses BGRA format; dunno why
	this->buffer[i] = packUnorm4x8(b, g, r, 1.0f);
}
