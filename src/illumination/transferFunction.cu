#include "hip/hip_runtime.h"
#include "transferFunction.h"
#include "consts.h"

#include <stdio.h>



__device__ float opacityFromGradient(const Vec3 &grad) {
    float gradMag = grad.length();
    float alpha = 1.0f - expf(-d_opacityK * gradMag);
    return alpha;
}

__device__ float opacitySigmoid(float val) {
    return 1.0f / (1.0f + expf(d_sigmoidTwo * (val - d_sigmoidOne)));
}

__device__ Color3 colorMap(float normalizedValues, const ColorStop stops[], int N) {
    // clamp to [0,1]
    normalizedValues = fminf(fmaxf(normalizedValues, 0.0f), 1.0f);

    // N stops => N-1 intervals
    for (int i = 0; i < N - 1; ++i) {
        float start = stops[i].pos;
        float end   = stops[i + 1].pos;

        if (normalizedValues >= start && normalizedValues <= end) {
            float localT = (normalizedValues - start) / (end - start);
            return interpolate(stops[i].color, stops[i + 1].color, localT);
        }
    }

    // fallback if something goes out of [0,1] or numerical issues
    return stops[N - 1].color;
}


// Transfer function
__device__ float4 transferFunction(float density, const Vec3& grad, const Point3& pos, const Vec3& rayDir) {
  
  // --------------------------- Sample the volume ---------------------------
  // TODO: Somehow pick if to use temp of speed normalization ... or pass extremas as params.
  float normDensity = (density - MIN_TEMP) / (MAX_TEMP - MIN_TEMP);
  // float normDensity = (density - MIN_SPEED) / (MAX_SPEED - MIN_SPEED);

  normDensity = clamp(normDensity, 0.0f, 1.0f);

  // --------------------------- Map density to color ---------------------------
  // TODO: Add a way to pick stops here
  Color3 baseColor = colorMap(normDensity, d_stopsPythonLike, lenStopsPythonLike);

  // TODO: This is a Gui select element
  // TODO: Add a way to pick different function for alpha
  float alpha = opacityFromGradient(grad);
  // alpha = 0.1f;
  alpha = opacitySigmoid(normDensity);
  // alpha = (1.0f - fabs(grad.normalize().dot(rayDir.normalize()))) * 0.8f + 0.2f;

  float alphaSample = density * alpha * 0.1;

  // --------------------------- Shading ---------------------------
  // Apply Phong
  Vec3 normal = -grad.normalize();
  Vec3 lightDir = (d_lightPos - pos).normalize();
  Vec3 viewDir  = -rayDir.normalize();
  Vec3 shadedColor = phongShading(normal, lightDir, viewDir, baseColor);  // TODO: Fix pixelated

  // Compose
  float4 result;
  result.x = shadedColor.x * alphaSample;
  result.y = shadedColor.y * alphaSample;
  result.z = shadedColor.z * alphaSample;
  result.w = alpha;

  // --------------------------- Silhouettes ---------------------------
  Vec3 N = grad.normalize();
  if (grad.length() > 0.2f && fabs(N.dot(viewDir)) < 0.02f) {
    result.x = 0.0f;
    result.y = 0.0f;
    result.z = 0.0f;
    result.w = 1.0f;
  }

  return result;
}
