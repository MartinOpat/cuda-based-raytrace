#include "hip/hip_runtime.h"
#include "Raycaster.h"

#include "hip/hip_runtime.h"
#include ""

#include "linalg/linalg.h"
#include "consts.h"
#include "transferFunction.h"
#include "cuda_error.h"

#include <iostream>
#include <hiprand/hiprand_kernel.h>

// TODO: instead of IMAGEWIDTH and IMAGEHEIGHT this should reflect the windowSize;
__global__ void raycastKernel(float* volumeData, FrameBuffer framebuffer, const int width, const int height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height) return;

    float accumR = 0.0f;
    float accumG = 0.0f;
    float accumB = 0.0f;
    float accumA = 1.0f * (float)d_samplesPerPixel;

    // Initialize random state for ray scattering
    hiprandState randState;
    hiprand_init(1234, px + py * width, 0, &randState);

    // Multiple samples per pixel
    for (int s = 0; s < d_samplesPerPixel; s++) {
        // Map to [-1, 1]
        float jitterU = (hiprand_uniform(&randState) - 0.5f) / width;
        float jitterV = (hiprand_uniform(&randState) - 0.5f) / height;
        float u = ((px + 0.5f + jitterU) / width ) * 2.0f - 1.0f;
        float v = ((py + 0.5f + jitterV) / height) * 2.0f - 1.0f;

        float tanHalfFov = tanf(fov * 0.5f);
        u *= tanHalfFov;
        v *= tanHalfFov;

        // Find ray direction
        Vec3 cameraRight = (d_cameraDir.cross(d_cameraUp)).normalize();
        d_cameraUp = (cameraRight.cross(d_cameraDir)).normalize();
        Vec3 rayDir = (d_cameraDir + cameraRight*u + d_cameraUp*v).normalize();

        // Intersect
        float tNear = 0.0f;
        float tFar  = 1e6f;
        auto intersectAxis = [&](float start, float dir, float minV, float maxV) {
            if (fabsf(dir) < epsilon) {
                // Ray parallel to axis. If outside min..max, no intersection.
                if (start < minV || start > maxV) {
                    tNear = 1e9f;
                    tFar  = -1e9f;
                }
            } else {
                float t0 = (minV - start) / dir;
                float t1 = (maxV - start) / dir;
                if (t0 > t1) {
                    float tmp = t0;
                    t0 = t1;
                    t1 = tmp;
                }
                if (t0 > tNear) tNear = t0;
                if (t1 < tFar ) tFar  = t1;
            }
        };

        intersectAxis(d_cameraPos.x, rayDir.x, 0.0f, (float)VOLUME_HEIGHT);
        intersectAxis(d_cameraPos.y, rayDir.y, 0.0f, (float)VOLUME_WIDTH);
        intersectAxis(d_cameraPos.z, rayDir.z, 0.0f, (float)VOLUME_DEPTH);

        if (tNear > tFar) {
          // No intersection -> Set to brackground color (multiply by d_samplesPerPixel because we divide by it later)
          accumR = d_backgroundColor.x * (float)d_samplesPerPixel;
          accumG = d_backgroundColor.y * (float)d_samplesPerPixel;
          accumB = d_backgroundColor.z * (float)d_samplesPerPixel;
          accumA = 1.0f * (float)d_samplesPerPixel;
          
        } else {
          if (tNear < 0.0f) tNear = 0.0f;

          float colorR = 0.0f, colorG = 0.0f, colorB = 0.0f;
          float alphaAccum = 0.0f;

          float t = tNear;  // Front to back
          while (t < tFar && alphaAccum < d_alphaAcumLimit) {
              Point3 pos = d_cameraPos + rayDir * t;

              // Convert to volume indices
              int ix = (int)roundf(pos.x);
              int iy = (int)roundf(pos.y);
              int iz = (int)roundf(pos.z);

              // Sample (pick appropriate method based on volume size) TODO: Consider adding a way to pick this in GUI (?)
              // float density = sampleVolumeNearest(volumeData, VOLUME_WIDTH, VOLUME_HEIGHT, VOLUME_DEPTH, ix, iy, iz);
              float density = sampleVolumeTrilinear(volumeData, VOLUME_WIDTH, VOLUME_HEIGHT, VOLUME_DEPTH, pos.x, pos.y, pos.z);

              // If density ~ 0, skip shading
              if (density > minAllowedDensity) {
                Vec3 grad = computeGradient(volumeData, VOLUME_WIDTH, VOLUME_HEIGHT, VOLUME_DEPTH, pos.x, pos.y, pos.z);
                float4 color = transferFunction(density, grad, pos, rayDir);  // This already returns the alpha-weighted color

                //Accumulate color, and alpha
                colorR = (1.0f - alphaAccum) * color.x + colorR;
                colorG = (1.0f - alphaAccum) * color.y + colorG;
                colorB = (1.0f - alphaAccum) * color.z + colorB;
                alphaAccum = (1 - alphaAccum) * color.w + alphaAccum;

              }


              t += stepSize;
          }


          // Calculate final colour
          accumR += colorR;
          accumG += colorG;
          accumB += colorB;
          accumA += alphaAccum;

          // Blend with background (for transparency)
          float leftover = 1.0 - alphaAccum;
          accumR = accumR + leftover * d_backgroundColor.x;
          accumG = accumG + leftover * d_backgroundColor.y;
          accumB = accumB + leftover * d_backgroundColor.z;
        }
    }


    // Average samples
    accumR /= (float)d_samplesPerPixel;
    accumG /= (float)d_samplesPerPixel;
    accumB /= (float)d_samplesPerPixel;
    accumA /= (float)d_samplesPerPixel;

    // Final colour
    framebuffer.writePixel(px, py, accumR, accumG, accumB, accumA);
}


Raycaster::Raycaster(hipGraphicsResource_t resources, int w, int h, float* data) {
	this->resources = resources;
	this->w = w;
	this->h = h;

	this->fb = new FrameBuffer(w, h);
  this->data = data;

	// camera_info = CameraInfo(Vec3(0.0f, 0.0f, 0.0f), Vec3(0.0f, 0.0f, 0.0f), 90.0f, (float) w, (float) h);
	// d_camera = thrust::device_new<Camera*>();

	check_cuda_errors(hipDeviceSynchronize());
}


void Raycaster::render() {
  check_cuda_errors(hipGraphicsMapResources(1, &this->resources));
	check_cuda_errors(hipGraphicsResourceGetMappedPointer((void**)&(this->fb->buffer), &(this->fb->buffer_size), resources));

  // FIXME: might not be the best parallelization configuration
	int tx = 8;
	int ty = 8;
	dim3 threadSize(this->w / tx + 1, this->h / ty + 1);
	dim3 blockSize(tx, ty);

  // TODO: pass camera info at some point
	// frame buffer is implicitly copied to the device each frame
  raycastKernel<<<threadSize, blockSize>>> (this->data, *this->fb, this->w, this->h);

  check_cuda_errors(hipGetLastError());
  check_cuda_errors(hipDeviceSynchronize());
  check_cuda_errors(hipGraphicsUnmapResources(1, &this->resources));
}


void Raycaster::resize(int w, int h) {
  this->w = w;
  this->h = h;

  delete this->fb;  
  this->fb = new FrameBuffer(w, h);

  // TODO: should be globals probably
	int tx = 8;
	int ty = 8;

	dim3 blocks(w / tx + 1, h / ty + 1);
	dim3 threads(tx, ty);

  check_cuda_errors(hipDeviceSynchronize());
}
