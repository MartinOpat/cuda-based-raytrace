#include "hip/hip_runtime.h"
#include "mat.h"
#include "consts.h"

#include <vector>
#include <algorithm>

using namespace std;

// Samples the voxel nearest to the given coordinates.
__device__ float sampleVolumeNearest(float* volumeData, const int volW, const int volH, const int volD, int vx, int vy, int vz) {
    // For boundary voxels - clamp to the boundary. 
    if (vx < 0) vx = 0;
    if (vy < 0) vy = 0;
    if (vz < 0) vz = 0;
    if (vx >= volH) vx = volH  - 1;
    if (vy >= volW) vy = volW - 1;
    if (vz >= volD) vz = volD  - 1;

    // x <-> height, y <-> width, z <-> depth
    int idx = vz * volW * volH + vx * volW + vy;
    return volumeData[idx];
}

// tri-linear interpolation - ready if necessary (but no visible improvement for full volume)
__device__ float sampleVolumeTrilinear(float* volumeData, const int volW, const int volH, const int volD, float fx, float fy, float fz) {
    int ix = (int)floorf(fx);
    int iy = (int)floorf(fy);
    int iz = (int)floorf(fz);

    // Clamp indices to valid range
    int ix1 = min(ix + 1, volH - 1);
    int iy1 = min(iy + 1, volW - 1);
    int iz1 = min(iz + 1, volD - 1);
    ix = max(ix, 0);
    iy = max(iy, 0);
    iz = max(iz, 0);

    // Compute weights
    float dx = fx - ix;
    float dy = fy - iy;
    float dz = fz - iz;

    // Sample values
    float c00 = sampleVolumeNearest(volumeData, volW, volH, volD, ix, iy, iz) * (1.0f - dx) +
                sampleVolumeNearest(volumeData, volW, volH, volD, ix1, iy, iz) * dx;
    float c10 = sampleVolumeNearest(volumeData, volW, volH, volD, ix, iy1, iz) * (1.0f - dx) +
                sampleVolumeNearest(volumeData, volW, volH, volD, ix1, iy1, iz) * dx;
    float c01 = sampleVolumeNearest(volumeData, volW, volH, volD, ix, iy, iz1) * (1.0f - dx) +
                sampleVolumeNearest(volumeData, volW, volH, volD, ix1, iy, iz1) * dx;
    float c11 = sampleVolumeNearest(volumeData, volW, volH, volD, ix, iy1, iz1) * (1.0f - dx) +
                sampleVolumeNearest(volumeData, volW, volH, volD, ix1, iy1, iz1) * dx;

    float c0 = c00 * (1.0f - dy) + c10 * dy;
    float c1 = c01 * (1.0f - dy) + c11 * dy;

    return c0 * (1.0f - dz) + c1 * dz;
}

__device__ Vec3 computeGradient(float* volumeData, const int volW, const int volH, const int volD, float fx, float fy, float fz) {
    // Compute gradient using central differencing with trilinear interpolation
    float hx = DLAT;  // x => height => lat
    float hy = DLON;  // y => width => lon
    float hz = DLEV;  // z => depth => alt
    
    // Default
    float dfdx = (sampleVolumeTrilinear(volumeData, volW, volH, volD, fx + hx, fy, fz) -
                  sampleVolumeTrilinear(volumeData, volW, volH, volD, fx - hx, fy, fz)) / (2.0f * hx);

    float dfdy = (sampleVolumeTrilinear(volumeData, volW, volH, volD, fx, fy + hy, fz) -
                  sampleVolumeTrilinear(volumeData, volW, volH, volD, fx, fy - hy, fz)) / (2.0f * hy);

    float dfdz = (sampleVolumeTrilinear(volumeData, volW, volH, volD, fx, fy, fz + hz) -
                  sampleVolumeTrilinear(volumeData, volW, volH, volD, fx, fy, fz - hz)) / (2.0f * hz);

    // // DEBUG (TODO: Delete) - Back to nearest
    // float dfdx = (sampleVolumeNearest(volumeData, volW, volH, volD, (int)roundf(fx + 1), (int)roundf(fy), (int)roundf(fz)) -
    //               sampleVolumeNearest(volumeData, volW, volH, volD, (int)roundf(fx - 1), (int)roundf(fy), (int)roundf(fz))) / (2.0f * hx);
    // float dfdy = (sampleVolumeNearest(volumeData, volW, volH, volD, (int)roundf(fx), (int)roundf(fy + 1), (int)roundf(fz)) -
    //               sampleVolumeNearest(volumeData, volW, volH, volD, (int)roundf(fx), (int)roundf(fy - 1), (int)roundf(fz))) / (2.0f * hy);
    // float dfdz = (sampleVolumeNearest(volumeData, volW, volH, volD, (int)roundf(fx), (int)roundf(fy), (int)roundf(fz + 1))  -
    //               sampleVolumeNearest(volumeData, volW, volH, volD, (int)roundf(fx), (int)roundf(fy), (int)roundf(fz - 1))) / (2.0f * hz);

    return Vec3::init(dfdx, dfdy, dfdz);
};

// TESTING: haven't tested this function at all tbh
__device__ unsigned int packUnorm4x8(float r, float g, float b, float a) {
  union {
	  unsigned char in[4];
	  uint out;
	} u;

  float len = sqrtf(r*r + g*g + b*b + a*a);

  // This is a Vec4 but i can't be bothered to make that its own struct/class; FIXME: maybe do that if we need to? From Martin: We could use a Vec4 for rgba too, but I don't feel like it either
  u.in[0] = round(r/len * 255.0f);
  u.in[1] = round(g/len * 255.0f);
  u.in[2] = round(b/len * 255.0f);
  u.in[3] = round(a/len * 255.0f);

	return u.out;
}

// Clamp a value between a min and max value
__device__ float clamp(float value, float min, float max) {
  return fmaxf(min, fminf(value, max));
}

// Normalize a float to the range [0, 1]
__device__ float normalize(float value, float min, float max) {
  return (value - min) / (max - min);
}