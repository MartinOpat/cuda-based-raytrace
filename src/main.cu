#include "hurricanedata/datareader.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <cmath>

int main() {
    std::string path = "data/MERRA2_400.inst6_3d_ana_Np.20120101.nc4";
    std::string variable = "U";
    auto arr = loadDataToDevice(path, variable);
    hipFreeArray(arr);

    return 0;
}
