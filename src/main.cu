#include "hip/hip_runtime.h"
#include "consts.h"
#include <hip/hip_runtime.h>
#include "gui/MainWindow.h"
#include "hurricanedata/datareader.h"
#include <iostream>
#include <vector>


static float* d_volume = nullptr;

void getTemperature(std::vector<float>& temperatureData, int idx = 0) {
    std::string path = "data/trimmed";
    // std::string path = "data";
    std::string variable = "T";
    DataReader dataReader(path, variable);
    size_t dataLength = dataReader.fileLength(idx);
    temperatureData.resize(dataLength);
    dataReader.loadFile(temperatureData.data(), idx);
}

void getSpeed(std::vector<float>& speedData, int idx = 0) {
    std::string path = "data/trimmed";
    // std::string path = "data";
    std::string varU = "U";
    std::string varV = "V";

    DataReader dataReaderU(path, varU);
    DataReader dataReaderV(path, varV);

    size_t dataLength = dataReaderU.fileLength(idx);
    speedData.resize(dataLength);
    std::vector<float> uData(dataLength);
    std::vector<float> vData(dataLength);

    dataReaderU.loadFile(uData.data(), idx);
    dataReaderV.loadFile(vData.data(), idx);

    for (int i = 0; i < dataLength; i++) {
        speedData[i] = sqrt(uData[i]*uData[i] + vData[i]*vData[i]);
    }
}


int main() {
  std::vector<float> data;
  getTemperature(data, 254); // 20121028
  // getSpeed(data, 254); // 20121028

  std::cout << "DATA size: " << data.size() << std::endl;

  // TODO: Eventually, we should not need to load the volume like this
  float* hostVolume = new float[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH];
  for (int i = 0; i < VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH; i++) {
    hostVolume[i] = data[i + 0*VOLUME_DEPTH*VOLUME_HEIGHT*VOLUME_WIDTH];
    // Discard missing values
    if (data[i + 0*VOLUME_DEPTH*VOLUME_HEIGHT*VOLUME_WIDTH] + epsilon >= infty) hostVolume[i] = -infty;
  }

  // Reverse the order of hostVolume - why is it upside down anyway?
  for (int i = 0; i < VOLUME_WIDTH; i++) {
    for (int j = 0; j < VOLUME_HEIGHT; j++) {
      for (int k = 0; k < VOLUME_DEPTH/2; k++) {
        float temp = hostVolume[i + j*VOLUME_WIDTH + k*VOLUME_WIDTH*VOLUME_HEIGHT];
        hostVolume[i + j*VOLUME_WIDTH + k*VOLUME_WIDTH*VOLUME_HEIGHT] = hostVolume[i + j*VOLUME_WIDTH + (VOLUME_DEPTH - 1 - k)*VOLUME_WIDTH*VOLUME_HEIGHT];
        hostVolume[i + j*VOLUME_WIDTH + (VOLUME_DEPTH - 1 - k)*VOLUME_WIDTH*VOLUME_HEIGHT] = temp;
      }
    }
  }

  // Allocate + copy data to GPU
  size_t volumeSize = sizeof(float) * VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH;
  hipMalloc((void**)&d_volume, volumeSize);
  hipMemcpy(d_volume, hostVolume, volumeSize, hipMemcpyHostToDevice);

  copyConstantsToDevice();
  // Create the GUI
  Window window(INITIAL_WINDOW_WIDTH, INITIAL_WINDOW_HEIGHT);
  int out = window.init(d_volume);

  // memory management
  hipFree(d_volume);
  delete[] hostVolume;
  return out;
}
