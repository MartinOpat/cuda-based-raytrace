#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include "consts.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include "gui/MainWindow.h"
#include "hurricanedata/datareader.h"
#include "illumination/illumination.h"
#include "img/handler.h"
#include <iostream>
#include "linalg/linalg.h" 
#include <vector>
#include <numeric>


static float* d_volume = nullptr;

// TODO: general
// * actual code for loading new data as the simulation progresses - right now its effectively a static image loader
// * save frames to file while running program -> then export to gif on close.
// * time controls - arbitrary skipping to specified point (would require some changes to gpubuffer) (could have)

void getTemperature(std::vector<float>& temperatureData, int idx = 0) {
    std::string path = "data/trimmed";
    // std::string path = "data";
    std::string variable = "T";
    DataReader dataReader(path, variable);
    size_t dataLength = dataReader.fileLength(idx);
    temperatureData.resize(dataLength);
    dataReader.loadFile(temperatureData.data(), idx);
}

void getSpeed(std::vector<float>& speedData, int idx = 0) {
    std::string path = "data/trimmed";
    // std::string path = "data";
    std::string varU = "U";
    std::string varV = "V";

    DataReader dataReaderU(path, varU);
    DataReader dataReaderV(path, varV);

    size_t dataLength = dataReaderU.fileLength(idx);
    speedData.resize(dataLength);
    std::vector<float> uData(dataLength);
    std::vector<float> vData(dataLength);

    dataReaderU.loadFile(uData.data(), idx);
    dataReaderV.loadFile(vData.data(), idx);

    for (int i = 0; i < dataLength; i++) {
        speedData[i] = sqrt(uData[i]*uData[i] + vData[i]*vData[i]);
    }
}


int main() {
  std::vector<float> data;
  getTemperature(data, 0);
  // getSpeed(data, 294);

  std::cout << "DATA size: " << data.size() << std::endl;

  // TODO: Eventually, we should not need to load the volume like this
  float* hostVolume = new float[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH];
  for (int i = 0; i < VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH; i++) {
    hostVolume[i] = data[i + 0*VOLUME_DEPTH*VOLUME_HEIGHT*VOLUME_WIDTH];
    // Discard missing values
    if (data[i + 0*VOLUME_DEPTH*VOLUME_HEIGHT*VOLUME_WIDTH] + epsilon >= infty) hostVolume[i] = -infty;
  }

  // Reverse the order of hostVolume - why is it upside down anyway?
  for (int i = 0; i < VOLUME_WIDTH; i++) {
    for (int j = 0; j < VOLUME_HEIGHT; j++) {
      for (int k = 0; k < VOLUME_DEPTH/2; k++) {
        float temp = hostVolume[i + j*VOLUME_WIDTH + k*VOLUME_WIDTH*VOLUME_HEIGHT];
        hostVolume[i + j*VOLUME_WIDTH + k*VOLUME_WIDTH*VOLUME_HEIGHT] = hostVolume[i + j*VOLUME_WIDTH + (VOLUME_DEPTH - 1 - k)*VOLUME_WIDTH*VOLUME_HEIGHT];
        hostVolume[i + j*VOLUME_WIDTH + (VOLUME_DEPTH - 1 - k)*VOLUME_WIDTH*VOLUME_HEIGHT] = temp;
      }
    }
  }

  // // Store the half-way up slice data into a file TODO: Remove this debug
  // std::ofstream myfile;
  // myfile.open("halfwayup.txt");
  // for (int i = 0; i < VOLUME_WIDTH; i++) {
  //   for (int j = 0; j < VOLUME_HEIGHT; j++) {
  //     myfile << hostVolume[i + j*VOLUME_WIDTH + VOLUME_DEPTH/2*VOLUME_WIDTH*VOLUME_HEIGHT] << " ";
  //   }
  //   myfile << std::endl;
  // }
  // myfile.close();

  // // Print min, max, avg., and median values TODO: Remove this debug
  // float minVal = *std::min_element(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH, [](float a, float b) {
  //   if (a <= epsilon) return false;
  //   if (b <= epsilon) return true;
  //   return a < b;
  // });
  // float maxVal = *std::max_element(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  // std::cout << "minVal: " << minVal << " maxVal: " << maxVal << std::endl;
  // // print min, max, avg., and median values <--- the code actually does not work when this snippet is enabled so probably TODO: Delete this later
  // std::sort(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  // float sum = std::accumulate(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH, 0.0f);
  // float avg = sum / (VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  // std::cout << "min: " << hostVolume[0] << " max: " << hostVolume[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH - 1] << " avg: " << avg << " median: " << hostVolume[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH / 2] << std::endl;

  // Allocate + copy data to GPU
  size_t volumeSize = sizeof(float) * VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH;
  hipMalloc((void**)&d_volume, volumeSize);
  hipMemcpy(d_volume, hostVolume, volumeSize, hipMemcpyHostToDevice);

  copyConstantsToDevice();
  // Create the GUI
  Window window(INITIAL_WINDOW_WIDTH, INITIAL_WINDOW_HEIGHT);
  int out = window.init(d_volume);

  // memory management
  hipFree(d_volume);
  delete[] hostVolume;
  return out;
}
