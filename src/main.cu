#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include "consts.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include "gui/MainWindow.h"
#include "hurricanedata/datareader.h"
#include "illumination/illumination.h"
#include "img/handler.h"
#include <iostream>
#include "linalg/linalg.h" 
#include <vector>
#include <numeric>


static float* d_volume = nullptr;

// TODO: general
// * pass camera_info to the raycasting function - updated according to glfw.
// * on that note, code for handling input (mouse movement certainly, possibly free input / 4 pre-coded views, q/esc to quit, space for pause (would be were the 'simple' render idea would come in))
// * very similarly - actual code for loading new data as the simulation progresses - right now its effectively a static image loader

void getTemperature(std::vector<float>& temperatureData, int idx = 0) {
    std::string path = "data/trimmed";
    // std::string path = "data";
    std::string variable = "T";
    DataReader dataReader(path, variable);
    size_t dataLength = dataReader.fileLength(idx);
    temperatureData.resize(dataLength);
    dataReader.loadFile(temperatureData.data(), idx);
}

void getSpeed(std::vector<float>& speedData, int idx = 0) {
    std::string path = "data/trimmed";
    // std::string path = "data";
    std::string varU = "U";
    std::string varV = "V";

    DataReader dataReaderU(path, varU);
    DataReader dataReaderV(path, varV);

    size_t dataLength = dataReaderU.fileLength(idx);
    speedData.resize(dataLength);
    std::vector<float> uData(dataLength);
    std::vector<float> vData(dataLength);

    dataReaderU.loadFile(uData.data(), idx);
    dataReaderV.loadFile(vData.data(), idx);

    for (int i = 0; i < dataLength; i++) {
        speedData[i] = sqrt(uData[i]*uData[i] + vData[i]*vData[i]);
    }
}


int main() {
  std::vector<float> data;
  getTemperature(data, 0);
  // getSpeed(data, 294);

  std::cout << "DATA size: " << data.size() << std::endl;

  // TODO: Eventually, we should not need to load the volume like this
  float* hostVolume = new float[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH];
  for (int i = 0; i < VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH; i++) {
    hostVolume[i] = data[i + 0*VOLUME_DEPTH*VOLUME_HEIGHT*VOLUME_WIDTH];
    // Discard missing values
    if (data[i + 0*VOLUME_DEPTH*VOLUME_HEIGHT*VOLUME_WIDTH] + epsilon >= infty) hostVolume[i] = -infty;
  }

  // Reverse the order of hostVolume - why is it upside down anyway?
  for (int i = 0; i < VOLUME_WIDTH; i++) {
    for (int j = 0; j < VOLUME_HEIGHT; j++) {
      for (int k = 0; k < VOLUME_DEPTH/2; k++) {
        float temp = hostVolume[i + j*VOLUME_WIDTH + k*VOLUME_WIDTH*VOLUME_HEIGHT];
        hostVolume[i + j*VOLUME_WIDTH + k*VOLUME_WIDTH*VOLUME_HEIGHT] = hostVolume[i + j*VOLUME_WIDTH + (VOLUME_DEPTH - 1 - k)*VOLUME_WIDTH*VOLUME_HEIGHT];
        hostVolume[i + j*VOLUME_WIDTH + (VOLUME_DEPTH - 1 - k)*VOLUME_WIDTH*VOLUME_HEIGHT] = temp;
      }
    }
  }

  // Store the half-way up slice data into a file TODO: Remove this debug
  std::ofstream myfile;
  myfile.open("halfwayup.txt");
  for (int i = 0; i < VOLUME_WIDTH; i++) {
    for (int j = 0; j < VOLUME_HEIGHT; j++) {
      myfile << hostVolume[i + j*VOLUME_WIDTH + VOLUME_DEPTH/2*VOLUME_WIDTH*VOLUME_HEIGHT] << " ";
    }
    myfile << std::endl;
  }
  myfile.close();

  // Print min, max, avg., and median values TODO: Remove this debug
  float minVal = *std::min_element(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH, [](float a, float b) {
    if (a <= epsilon) return false;
    if (b <= epsilon) return true;
    return a < b;
  });
  float maxVal = *std::max_element(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  std::cout << "minVal: " << minVal << " maxVal: " << maxVal << std::endl;
  // // print min, max, avg., and median values <--- the code actually does not work when this snippet is enabled so probably TODO: Delete this later
  // std::sort(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  // float sum = std::accumulate(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH, 0.0f);
  // float avg = sum / (VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  // std::cout << "min: " << hostVolume[0] << " max: " << hostVolume[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH - 1] << " avg: " << avg << " median: " << hostVolume[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH / 2] << std::endl;

  // Allocate + copy data to GPU
  size_t volumeSize = sizeof(float) * VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH;
  hipMalloc((void**)&d_volume, volumeSize);
  hipMemcpy(d_volume, hostVolume, volumeSize, hipMemcpyHostToDevice);

  // Allocate framebuffer
  // unsigned char* d_framebuffer;
  // size_t fbSize = IMAGE_WIDTH * IMAGE_HEIGHT * 3 * sizeof(unsigned char);
  // hipMalloc((void**)&d_framebuffer, fbSize);
  // hipMemset(d_framebuffer, 0, fbSize);

  // Copy external constants from consts.h to cuda
  copyConstantsToDevice();

  // NOTE: this is done within the rayTracer class
  // // Launch kernel
  // dim3 blockSize(16, 16);  
  // dim3 gridSize((IMAGE_WIDTH + blockSize.x - 1)/blockSize.x,
  //               (IMAGE_HEIGHT + blockSize.y - 1)/blockSize.y);
  //
  // raycastKernel<<<gridSize, blockSize>>>(
  //     d_volume,
  //     d_framebuffer
  // );
  // hipDeviceSynchronize();

  Window window(IMAGE_WIDTH, IMAGE_HEIGHT);
  int out = window.init(d_volume);

  hipFree(d_volume);

  // // Copy framebuffer back to CPU
  // unsigned char* hostFramebuffer = new unsigned char[IMAGE_WIDTH * IMAGE_HEIGHT * 3];
  // hipMemcpy(hostFramebuffer, d_framebuffer, fbSize, hipMemcpyDeviceToHost);
  //
  // // Export image
  // saveImage("output.ppm", hostFramebuffer, IMAGE_WIDTH, IMAGE_HEIGHT);
  //
  // // Cleanup //TODO: cleanup properly
  delete[] hostVolume;
  // delete[] hostFramebuffer;
  // hipFree(d_volume);
  // hipFree(d_framebuffer);
  //
  // std::cout << "Phong-DVR rendering done. Image saved to output.ppm" << std::endl;
  // return 0;
  return out;
}
