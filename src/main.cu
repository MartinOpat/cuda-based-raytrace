#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include "consts.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include "gui/MainWindow.h"
#include "hurricanedata/datareader.h"
#include "illumination/illumination.h"
#include "img/handler.h"
#include <iostream>
#include "linalg/linalg.h" 
#include <vector>
#include <numeric>


static float* d_volume = nullptr;

// TODO: general
// * pass camera_info to the raycasting function - updated according to glfw.
// * on that note, code for handling input (mouse movement certainly, possibly free input / 4 pre-coded views, q/esc to quit, space for pause (would be were the 'simple' render idea would come in))
// * very similarly - actual code for loading new data as the simulation progresses - right now its effectively a static image loader

void getTemperature(std::vector<float>& temperatureData, int idx = 0) {
    std::string path = "data/trimmed";
    // std::string path = "data";
    std::string variable = "T";
    DataReader dataReader(path, variable);
    size_t dataLength = dataReader.fileLength(idx);
    temperatureData.resize(dataLength);
    dataReader.loadFile(temperatureData.data(), idx);
}

void getSpeed(std::vector<float>& speedData, int idx = 0) {
    std::string path = "data/trimmed";
    // std::string path = "data";
    std::string varU = "U";
    std::string varV = "V";

    DataReader dataReaderU(path, varU);
    DataReader dataReaderV(path, varV);

    size_t dataLength = dataReaderU.fileLength(idx);
    speedData.resize(dataLength);
    std::vector<float> uData(dataLength);
    std::vector<float> vData(dataLength);

    dataReaderU.loadFile(uData.data(), idx);
    dataReaderV.loadFile(vData.data(), idx);

    for (int i = 0; i < dataLength; i++) {
        speedData[i] = sqrt(uData[i]*uData[i] + vData[i]*vData[i]);
    }
}


int main() {
  std::vector<float> data;
  getTemperature(data, 0);
  // getSpeed(data, 294);

  std::cout << "DATA size: " << data.size() << std::endl;

  // TODO: Eveontually remove debug below (i.e., eliminate for-loop etc.)
  // Generate debug volume data
  float* hostVolume = new float[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH];
  // generateVolume(hostVolume, VOLUME_WIDTH, VOLUME_HEIGHT, VOLUME_DEPTH);
  int inftyCount=0;
  for (int i = 0; i < VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH; i++) {
    // Discard temperatures above a small star (supposedly, missing temperature values)
    hostVolume[i] = data[i + 0*VOLUME_DEPTH*VOLUME_HEIGHT*VOLUME_WIDTH];
    if (data[i + 0*VOLUME_DEPTH*VOLUME_HEIGHT*VOLUME_WIDTH] + epsilon >= infty) {hostVolume[i] = -infty; inftyCount++;}
  }
  std::cout << "inftyCount: " << inftyCount << std::endl;

  // Reverse the order of hostVolume
  for (int i = 0; i < VOLUME_WIDTH; i++) {
    for (int j = 0; j < VOLUME_HEIGHT; j++) {
      for (int k = 0; k < VOLUME_DEPTH/2; k++) {
        float temp = hostVolume[i + j*VOLUME_WIDTH + k*VOLUME_WIDTH*VOLUME_HEIGHT];
        hostVolume[i + j*VOLUME_WIDTH + k*VOLUME_WIDTH*VOLUME_HEIGHT] = hostVolume[i + j*VOLUME_WIDTH + (VOLUME_DEPTH - 1 - k)*VOLUME_WIDTH*VOLUME_HEIGHT];
        hostVolume[i + j*VOLUME_WIDTH + (VOLUME_DEPTH - 1 - k)*VOLUME_WIDTH*VOLUME_HEIGHT] = temp;
      }
    }
  }


  // Store the half-way up slice data into a file
  std::ofstream myfile;
  myfile.open("halfwayup.txt");
  for (int i = 0; i < VOLUME_WIDTH; i++) {
    for (int j = 0; j < VOLUME_HEIGHT; j++) {
      myfile << hostVolume[i + j*VOLUME_WIDTH + VOLUME_DEPTH/2*VOLUME_WIDTH*VOLUME_HEIGHT] << " ";
    }
    myfile << std::endl;
  }
  myfile.close();

  float minVal = *std::min_element(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH, [](float a, float b) {
    if (a <= epsilon) return false;
    if (b <= epsilon) return true;
    return a < b;
  });
  float maxVal = *std::max_element(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  std::cout << "minVal: " << minVal << " maxVal: " << maxVal << std::endl;

  // Min-max normalization TODO: Decide whether to keep the normalization here but probably not
  // Normalize to [0, 1]
  // Temperature: min: 0 max: 1 avg: 0.776319 median: 0.790567
  // Speed: min: 0 max: 1 avg: 0.132117 median: 0.0837869
  // for (int i = 0; i < VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH; i++) {
  //   hostVolume[i] = (hostVolume[i] - minVal) / (maxVal - minVal);
  // }

  // // print min, max, avg., and median values <--- the code actually does not work when this snippet is enabled so probably TODO: Delete this later
  // std::sort(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  // float sum = std::accumulate(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH, 0.0f);
  // float avg = sum / (VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
  // std::cout << "min: " << hostVolume[0] << " max: " << hostVolume[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH - 1] << " avg: " << avg << " median: " << hostVolume[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH / 2] << std::endl;

  // Allocate + copy data to GPU
  size_t volumeSize = sizeof(float) * VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH;
  hipMalloc((void**)&d_volume, volumeSize);
  hipMemcpy(d_volume, hostVolume, volumeSize, hipMemcpyHostToDevice);

  // Allocate framebuffer
  // unsigned char* d_framebuffer;
  // size_t fbSize = IMAGE_WIDTH * IMAGE_HEIGHT * 3 * sizeof(unsigned char);
  // hipMalloc((void**)&d_framebuffer, fbSize);
  // hipMemset(d_framebuffer, 0, fbSize);

  // Copy external constants from consts.h to cuda
  copyConstantsToDevice();

  // NOTE: this is done within the rayTracer class
  // // Launch kernel
  // dim3 blockSize(16, 16);  
  // dim3 gridSize((IMAGE_WIDTH + blockSize.x - 1)/blockSize.x,
  //               (IMAGE_HEIGHT + blockSize.y - 1)/blockSize.y);
  //
  // raycastKernel<<<gridSize, blockSize>>>(
  //     d_volume,
  //     d_framebuffer
  // );
  // hipDeviceSynchronize();

  Window window(IMAGE_WIDTH, IMAGE_HEIGHT);
  int out = window.init(d_volume);

  hipFree(d_volume);

  // // Copy framebuffer back to CPU
  // unsigned char* hostFramebuffer = new unsigned char[IMAGE_WIDTH * IMAGE_HEIGHT * 3];
  // hipMemcpy(hostFramebuffer, d_framebuffer, fbSize, hipMemcpyDeviceToHost);
  //
  // // Export image
  // saveImage("output.ppm", hostFramebuffer, IMAGE_WIDTH, IMAGE_HEIGHT);
  //
  // // Cleanup //TODO: cleanup properly
  delete[] hostVolume;
  // delete[] hostFramebuffer;
  // hipFree(d_volume);
  // hipFree(d_framebuffer);
  //
  // std::cout << "Phong-DVR rendering done. Image saved to output.ppm" << std::endl;
  // return 0;
  return out;
}
