#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

#include "hurricanedata/datareader.h"
#include "linalg/linalg.h" 
#include "objs/sphere.h"
#include "img/handler.h"
#include "consts.h"


__constant__ int d_volumeWidth;
__constant__ int d_volumeHeight;
__constant__ int d_volumeDepth;

static float* d_volume = nullptr;

// ----------------------------------------------------------------------------------------------------
__device__ Vec3 phongShading(const Vec3& normal, const Vec3& lightDir, const Vec3& viewDir, const Vec3& baseColor) {
    Vec3 ambient = baseColor * ambientStrength;
    double diff = fmax(normal.dot(lightDir), 0.0);
    Vec3 diffuse = baseColor * (diffuseStrength * diff);

    Vec3 reflectDir = (normal * (2.0 * normal.dot(lightDir)) - lightDir).normalize();
    double spec = pow(fmax(viewDir.dot(reflectDir), 0.0), shininess);
    Vec3 specular = Vec3(1.0, 1.0, 1.0) * (specularStrength * spec);

    return ambient + diffuse + specular;
}

// Raycast + phong
__global__ void raycastKernel(float*  volumeData, unsigned char* framebuffer, int imageWidth, int imageHeight, Vec3 cameraPos, Vec3 cameraDir, Vec3 cameraUp, float fov, float stepSize, Vec3 lightPos) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= imageWidth || py >= imageHeight) return;

    float accumR = 0.0f;
    float accumG = 0.0f;
    float accumB = 0.0f;

    // Multiple samples per pixel
    for (int s = 0; s < SAMPLES_PER_PIXEL; s++) {
        // Map to [-1, 1]
        float u = ((px + 0.5f) / imageWidth ) * 2.0f - 1.0f;
        float v = ((py + 0.5f) / imageHeight) * 2.0f - 1.0f;

        // TODO: Move this (and all similar transformation code) to its own separate file
        float tanHalfFov = tanf(fov * 0.5f);
        u *= tanHalfFov;
        v *= tanHalfFov;

        // Find ray direction
        Vec3 cameraRight = (cameraDir.cross(cameraUp)).normalize();
        cameraUp = (cameraRight.cross(cameraDir)).normalize();
        Vec3 rayDir = (cameraDir + cameraRight*u + cameraUp*v).normalize();

        // Intersect (for simplicity just a 3D box from 0 to 1 in all dimensions) - TODO: Think about whether this is the best way to do this
        float tNear = 0.0f;
        float tFar  = 1e6f;
        auto intersectAxis = [&](float start, float dirVal) {
            if (fabsf(dirVal) < epsilon) {
                if (start < 0.f || start > 1.f) {
                    tNear = 1e9f;
                    tFar  = -1e9f;
                }
            } else {
                float t0 = (0.0f - start) / dirVal;
                float t1 = (1.0f - start) / dirVal;
                if (t0>t1) { 
                    float tmp=t0; 
                    t0=t1; 
                    t1=tmp; 
                }
                if (t0>tNear) tNear = t0;
                if (t1<tFar ) tFar  = t1;
            }
        };

        intersectAxis(cameraPos.x, rayDir.x);
        intersectAxis(cameraPos.y, rayDir.y);
        intersectAxis(cameraPos.z, rayDir.z);

        if (tNear > tFar) continue;  // No intersectionn
        if (tNear < 0.0f) tNear = 0.0f;

        float colorR = 0.0f, colorG = 0.0f, colorB = 0.0f;
        float alphaAccum = 0.0f;

        float tCurrent = tNear;
        while (tCurrent < tFar && alphaAccum < alphaAcumLimit) {
            Vec3 pos = cameraPos + rayDir * tCurrent;

            // Convert to volume indices
            float fx = pos.x * (d_volumeWidth  - 1);
            float fy = pos.y * (d_volumeHeight - 1);
            float fz = pos.z * (d_volumeDepth  - 1);
            int ix = (int)roundf(fx);
            int iy = (int)roundf(fy);
            int iz = (int)roundf(fz);

            // Sample
            float density = sampleVolumeNearest(volumeData, d_volumeWidth, d_volumeHeight, d_volumeDepth, ix, iy, iz);

            // Basic transfer function. TODO: Move to a separate file, and then improve
            float alphaSample = density * 0.1f;
            // float alphaSample = 1.0f - expf(-density * 0.1f);
            Vec3 baseColor = Vec3(density, 0.1f*density, 1.f - density);  // TODO: Implement a proper transfer function

            // If density ~ 0, skip shading
            if (density > minAllowedDensity) {
                Vec3 grad = computeGradient(volumeData, d_volumeWidth, d_volumeHeight, d_volumeDepth, ix, iy, iz);
                Vec3 normal = -grad.normalize();

                Vec3 lightDir = (lightPos - pos).normalize();
                Vec3 viewDir  = -rayDir.normalize();

                // Apply Phong
                Vec3 shadedColor = phongShading(normal, lightDir, viewDir, baseColor);

                // Compose
                colorR     += (1.0f - alphaAccum) * shadedColor.x * alphaSample;
                colorG     += (1.0f - alphaAccum) * shadedColor.y * alphaSample;
                colorB     += (1.0f - alphaAccum) * shadedColor.z * alphaSample;
                alphaAccum += (1.0f - alphaAccum) * alphaSample;
            }

            tCurrent += stepSize;
        }

        accumR += colorR;
        accumG += colorG;
        accumB += colorB;
    }

    // Average samples
    accumR /= (float)SAMPLES_PER_PIXEL;
    accumG /= (float)SAMPLES_PER_PIXEL;
    accumB /= (float)SAMPLES_PER_PIXEL;

    // Final colour
    int fbIndex = (py * imageWidth + px) * 3;
    framebuffer[fbIndex + 0] = (unsigned char)(fminf(accumR, 1.f) * 255);
    framebuffer[fbIndex + 1] = (unsigned char)(fminf(accumG, 1.f) * 255);
    framebuffer[fbIndex + 2] = (unsigned char)(fminf(accumB, 1.f) * 255);
}

void getTemperature(std::vector<float>& temperatureData, int idx = 0) {
    std::string path = "data/trimmed";
    std::string variable = "T";
    DataReader dataReader(path, variable);
    size_t dataLength = dataReader.fileLength(idx);
    temperatureData.resize(dataLength);
    dataReader.loadFile(temperatureData.data(), idx);
}

void getSpeed(std::vector<float>& speedData, int idx = 0) {
    std::string path = "data/trimmed";
    std::string varU = "U";
    std::string varV = "V";

    DataReader dataReaderU(path, varU);
    DataReader dataReaderV(path, varV);

    size_t dataLength = dataReaderU.fileLength(idx);
    speedData.resize(dataLength);
    std::vector<float> uData(dataLength);
    std::vector<float> vData(dataLength);

    dataReaderU.loadFile(uData.data(), idx);
    dataReaderV.loadFile(vData.data(), idx);

    for (int i = 0; i < dataLength; i++) {
        speedData[i] = sqrt(uData[i]*uData[i] + vData[i]*vData[i]);
    }
}

int main(int argc, char** argv) {
    std::vector<float> data;
    // getTemperature(data);
    getSpeed(data);


    // Generate debug volume data
    float* hostVolume = new float[VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH];
    // generateVolume(hostVolume, VOLUME_WIDTH, VOLUME_HEIGHT, VOLUME_DEPTH);
    for (int i = 0; i < VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH; i++) {  // TODO: This is technically an unnecessary artifact of the old code taking in a float* instead of a std::vector
        // Discard temperatures above a small star (supposedly, missing temperature values)
        hostVolume[i] = data[i];
        if (data[i] + epsilon >= infty) hostVolume[i] = 0.0f;
    }

        // Min-max normalization
    float minVal = *std::min_element(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
    float maxVal = *std::max_element(hostVolume, hostVolume + VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH);
    for (int i = 0; i < VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH; i++) {
        hostVolume[i] = (hostVolume[i] - minVal) / (maxVal - minVal);
    }

    // Allocate + copy data to GPU
    size_t volumeSize = sizeof(float) * VOLUME_WIDTH * VOLUME_HEIGHT * VOLUME_DEPTH;
    hipMalloc((void**)&d_volume, volumeSize);
    hipMemcpy(d_volume, hostVolume, volumeSize, hipMemcpyHostToDevice);

    int w = VOLUME_WIDTH, h = VOLUME_HEIGHT, d = VOLUME_DEPTH;
    hipMemcpyToSymbol(HIP_SYMBOL(d_volumeWidth),  &w, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_volumeHeight), &h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_volumeDepth),  &d, sizeof(int));

    // Allocate framebuffer
    unsigned char* d_framebuffer;
    size_t fbSize = IMAGE_WIDTH * IMAGE_HEIGHT * 3 * sizeof(unsigned char);
    hipMalloc((void**)&d_framebuffer, fbSize);
    hipMemset(d_framebuffer, 0, fbSize);

    // Camera and Light
    Vec3 cameraPos(-0.7, -1.0, -2.0);
    Vec3 cameraDir(0.4, 0.6, 1.0);
    Vec3 cameraUp(0.0, 1.0, 0.0);
    float fov = 60.0f * (M_PI / 180.0f);
    float stepSize = 0.002f;
    Vec3 lightPos(1.5, 2.0, -1.0);

    // Launch kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((IMAGE_WIDTH + blockSize.x - 1)/blockSize.x,
                  (IMAGE_HEIGHT + blockSize.y - 1)/blockSize.y);

    raycastKernel<<<gridSize, blockSize>>>(
        d_volume,
        d_framebuffer,
        IMAGE_WIDTH,
        IMAGE_HEIGHT,
        cameraPos,
        cameraDir.normalize(),
        cameraUp.normalize(),
        fov,
        stepSize,
        lightPos
    );
    hipDeviceSynchronize();

    // Copy framebuffer back to CPU
    unsigned char* hostFramebuffer = new unsigned char[IMAGE_WIDTH * IMAGE_HEIGHT * 3];
    hipMemcpy(hostFramebuffer, d_framebuffer, fbSize, hipMemcpyDeviceToHost);

    // Export image
    saveImage("output.ppm", hostFramebuffer, IMAGE_WIDTH, IMAGE_HEIGHT);

    // Cleanup
    delete[] hostVolume;
    delete[] hostFramebuffer;
    hipFree(d_volume);
    hipFree(d_framebuffer);

    std::cout << "Phong-DVR rendering done. Image saved to output.ppm" << std::endl;
    return 0;
}

// // gpu-buffer-handler branch main
// #include "hurricanedata/fielddata.h"
// #include "hurricanedata/gpubufferhandler.h"
// #include "hurricanedata/datareader.h"
// #include "hurricanedata/gpubuffer.h"

// #include <hip/hip_runtime.h>
// #include <>
// #include <iostream>
// #include <cmath>
// #include <memory>
// #include <iomanip> 

// __global__ void middleOfTwoValues(float *ans, const FieldMetadata &fmd, FieldData fd) {
//     float xi = getVal(fmd, fd, 0, 20, 100, 100);
//     float yi = getVal(fmd, fd, 1, 20, 100, 100);
//     *ans = (xi+yi)/2;
// }

// int main() {
//     // std::string path = "data/atmosphere_MERRA-wind-speed[179253532]";
//     std::string path = "data/trimmed";

//     std::string variable = "T";

//     DataReader dataReader{path, variable};

//     std::cout << "created datareader\n";

//     GPUBuffer buffer (dataReader);

//     std::cout << "created buffer\n";

//     GPUBufferHandler bufferHandler(buffer);

//     float *ptr_test_read;
//     hipMallocManaged(&ptr_test_read, sizeof(float));

//     std::cout << "created buffer handler\n";
//     for (int i = 0; i < 10; i++) {
//         FieldData fd = bufferHandler.nextFieldData();

//         middleOfTwoValues<<<1, 1>>>(ptr_test_read, *bufferHandler.fmd, fd);

//         hipDeviceSynchronize();
//         std::cout << "ptr_test_read = " << std::fixed << std::setprecision(6) << *ptr_test_read << "\n";
//     }
    
//     // TODO: measure data transfer time in this example code.
//     hipFree(ptr_test_read);
//     return 0;
// }



