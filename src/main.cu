#include "hip/hip_runtime.h"
// #include "hurricanedata/fielddata.h"
#include "hurricanedata/gpubufferhandler.h"
#include "hurricanedata/datareader.h"
#include "hurricanedata/gpubuffer.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <cmath>
#include <memory>
#include <iomanip> 

__global__ void getSingleValue(float *ans, const FieldMetadata &fmd, FieldData fd) {
    float xi = getVal(fmd, fd, 1, 20, 100, 100);
    *ans = xi;
}

int main() {
    std::string path = "data/atmosphere_MERRA-wind-speed[179253532]";

    std::string variable = "T";

    // std::unique_ptr<DataReader> dataReader = std::make_unique<DataReader>(path, variable);
    DataReader dataReader{path, variable};

    std::cout << "created datareader\n";

    GPUBuffer buffer (dataReader);

    std::cout << "created buffer\n";

    GPUBufferHandler bufferHandler(buffer);

    float *ptr_test_read;
    hipMallocManaged(&ptr_test_read, sizeof(float));

    std::cout << "created buffer handler\n";
    for (int i = 0; i < 10; i++) {
        FieldData fd = bufferHandler.nextFieldData();

        getSingleValue<<<1, 1>>>(ptr_test_read, *bufferHandler.fmd, fd);

        hipDeviceSynchronize();

        std::cout << "ptr_test_read = " << std::fixed << std::setprecision(6) << *ptr_test_read << "\n";
    }
    
    // TODO: Write an example loop using buffering and measure it.

    // TODO: Free data properly in FieldData (maybe make an iterator)
    hipFree(ptr_test_read);
    return 0;
}
