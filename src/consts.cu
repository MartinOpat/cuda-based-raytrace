#include "consts.h"

__device__ Point3 d_cameraPos;
__device__ Vec3 d_cameraDir;
__device__ Vec3 d_cameraUp;
__device__ Point3 d_lightPos;

Point3 h_cameraPos = Point3::init(-300.0f, 200.0f, -300.0f);
Vec3 center = Vec3::init((float)VOLUME_WIDTH/2.0f, (float)VOLUME_HEIGHT/2.0f, (float)VOLUME_DEPTH/2.0f);
Vec3 h_cameraDir = (center - h_cameraPos).normalize();
Vec3 h_cameraUp = Vec3::init(0.0, 1.0, 0.0).normalize();
Point3 h_lightPos = Point3::init(1.5, 2.0, -1.0);

void copyConstantsToDevice() {
    hipMemcpyToSymbol(HIP_SYMBOL(d_cameraPos), &h_cameraPos, sizeof(Point3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_cameraDir), &h_cameraDir, sizeof(Vec3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_cameraUp), &h_cameraUp, sizeof(Vec3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_lightPos), &h_lightPos, sizeof(Point3));
}
