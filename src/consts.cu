#include "consts.h"

// ----------------------- Colour mapping -----------------------
__constant__ ColorStop d_stopsPythonLike[5];
__constant__ ColorStop d_stopsGrayscale[2];
__constant__ ColorStop d_stopsBluePurleRed[3];

const ColorStop h_stopsPythonLike[] = {
        { 0.0f, Color3::init(0.2298057f, 0.29871797f, 0.75368315f) }, // Dark Blue
        { 0.25f, Color3::init(0.23437708f, 0.30554173f, 0.75967953f) }, // Mid Blue
        { 0.5f, Color3::init(0.27582712f, 0.36671692f, 0.81255294f) }, // White
        { 0.75f, Color3::init(0.79606387f, 0.84869321f, 0.93347147f) }, // Light Orange
        { 1.0f, Color3::init(0.70567316f, 0.01555616f, 0.15023281f) }  // Red
};

const ColorStop h_stopsGrayscale[] = {
        { 0.0f, Color3::init(0.0f, 0.0f, 0.0f) },  // No colour
        { 1.0f, Color3::init(1.0f, 1.0f, 1.0f) }   // White
};

const ColorStop h_stopsBluePurleRed[] = {
        { 0.0f, Color3::init(0.0f, 0.0f, 1.0f) },   // deep blue
        { 0.5f, Color3::init(0.5f, 0.0f, 0.5f) },   // purple
        { 1.0f, Color3::init(1.0f, 0.0f, 0.0f) }    // deep red
};

// ----------------------- Camera and Light -----------------------

__device__ Point3 d_cameraPos;
__device__ Vec3 d_cameraDir;
__device__ Vec3 d_cameraUp;
__device__ Point3 d_lightPos;

// Point3 h_cameraPos = Point3::init(300.0f, 200.0f, -700.0f);  // Camera for full data set
Point3 h_cameraPos = Point3::init(50.0f, -50.0f, -75.0f);  // Camera for partially trimmed data set (TODO: Probably upside down atm)
Vec3 center = Vec3::init((float)VOLUME_WIDTH/2.0f, (float)VOLUME_HEIGHT/2.0f, (float)VOLUME_DEPTH/2.0f);
Vec3 h_cameraDir = (center - h_cameraPos).normalize();
Vec3 h_cameraUp = Vec3::init(0.0, 0.0, 1.0).normalize();
Point3 h_lightPos = Point3::init(1.5, 2.0, -1.0);


// Copy the above values to the device
void copyConstantsToDevice() {
    // ----------------------- Colour mapping -----------------------
    hipMemcpyToSymbol(HIP_SYMBOL(d_stopsPythonLike), h_stopsPythonLike, sizeof(h_stopsPythonLike));
    hipMemcpyToSymbol(HIP_SYMBOL(d_stopsGrayscale), h_stopsGrayscale, sizeof(h_stopsGrayscale));
    hipMemcpyToSymbol(HIP_SYMBOL(d_stopsBluePurleRed), h_stopsBluePurleRed, sizeof(h_stopsBluePurleRed));


    // ----------------------- Camera and Light -----------------------
    hipMemcpyToSymbol(HIP_SYMBOL(d_cameraPos), &h_cameraPos, sizeof(Point3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_cameraDir), &h_cameraDir, sizeof(Vec3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_cameraUp), &h_cameraUp, sizeof(Vec3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_lightPos), &h_lightPos, sizeof(Point3));
}
