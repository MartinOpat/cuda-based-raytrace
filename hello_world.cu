#include <iostream>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU!\n");
}

int main() {
    hello_from_gpu<<<1, 1>>>();

    hipDeviceSynchronize();

    // Reset device
    hipDeviceReset();
    return 0;
}
