#include <iostream>
#include <hip/hip_runtime.h>

#define cudaCheckError() {                                      \
    hipError_t e = hipGetLastError();                         \
    if (e != hipSuccess) {                                     \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__,    \
               hipGetErrorString(e));                          \
        exit(EXIT_FAILURE);                                     \
    }                                                           \
}

__global__ void hello_from_gpu() {
    printf("Hello from GPU!\n");
}

int main() {
    hello_from_gpu<<<1, 1>>>();
    cudaCheckError();

    hipDeviceSynchronize();
    cudaCheckError();

    // Reset device
    hipDeviceReset();
    return 0;
}
